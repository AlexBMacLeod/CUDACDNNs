#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include "Linear.h"
#include "../include/nnKernel.cuh"

#define TILE_SIZE 32
#define CHECK_ERROR(call) { \
	hipError_t err = call; \
	if (err != hipSuccess) { \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(err); \
	} \
}

__global__ void vecxvec_kernel(const float* __restrict__ d_m, const float* __restrict__ d_x, float * __restrict__ d_p,
    const unsigned int nRows, const unsigned int nCols)
{
    const unsigned int tid = blockDimx.x * blockIdx.x + threadIdx.x;
    __shared__ float xds[TILE_SIZE];
    float pval = 0.0;

    #pragma unroll
    for(unsigned int m = 0; m < ((nCols + TILE_SIZE -1)/TILE_SIZE); m++)
    {
        if((m * TILE_SIZE + threadIdx.x) < nCols)
        {
            xds[threadIdx.x] = d_x[threadIdx.x + m * TILE_SIZE];
        } else xds[threadIdx.x] = 0.f;
    }
    __syncthreads();

    if ((Row<Width) && (Col<Width)) p[tid] = Pvalue;

    #pragma unroll
    for(unsigned int e = 0; e < TILE_SIZE; e++)
    {
        pval += d_m[t + (e + TILE_SIZE *m) * nRows] * xds[e];
    }
    
    if ((Row<Width) && (Col<Width)) P[Row*Width+Col] = Pvalue;
}


__global__ void matvec_kernel(const float* __restrict__ d_M, const float* __restrict__ d_x, float * __restrict__ d_p, 
    const unsigned int nRows, const unsigned int nCols)
{
    const unsigned int tid = blockDimx.x * blockIdx.x + threadIdx.x;
    __shared__ float xds[TILE_SIZE];
    float pval = 0.0;

    #pragma unroll
    for(unsigned int m = 0; m < ((nCols + TILE_SIZE -1)/TILE_SIZE); m++)
    {
        if((m * TILE_SIZE + threadIdx.x) < nCols)
        {
            xds[threadIdx.x] = d_x[threadIdx.x + m * TILE_SIZE];
        } else xds[threadIdx.x] = 0.f;
    
        __syncthreads();

        #pragma unroll
        for(unsigned int e = 0; e < TILE_SIZE; e++)
        {
            pval += d_M[tid + (e + TILE_SIZE *m) * nRows] * xds[e];
        }
    }
}

__global__ void transpose_kernel(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCK_DIM][BLOCK_DIM+1];


    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

void matrixVector(struct linearLayer* layer, float *input, float *output)
{
    float *d_Out;
    float *d_Vec;
    float *d_Matrix;
    int sizeMatrix = layer->in * layer->out * sizeof(float);
    int sizeInVec = layer->in * sizeof(float);
    int sizeOutVec = layer->out * sizeof(float);

    CHECK_ERROR(hipMalloc((**void)&d_Matrix, sizeMatrix));
    CHECK_ERROR(hipMalloc((**void)&d_Vec, sizeInVec));
    CHECK_ERROR(hipMalloc((**void)&d_Out, sizeOutVec));

    dim3 dimGrid(ceil(layer->in/32.0), ceil(layer->out/32.0), 1);
    dim3 dimBlock(32.0, 32.0, 1);

    matvec_kernel<<<dimGrid, dimBlock>>>(d_Matrix, d_Vec, d_Out, layer->in, layer->out);

    cudaMemCpy(d_Out, output, sizeOutVec, cudaMemCpyDeviceToHost);

    hipFree(d_Out);
    hipFree(d_Vec);
    hipFree(d_Matrix);
}

void transpose(struct linearLayer* layer)
{
    (float *odata, float *idata, int width, int height)
}

void forwardPass(float *inputRow, layer hiddenLayers[],
    float *yhat, int numHidden)
{
    float *d_Out;
    float *d_Vec;
    float *d_Matrix;
    int sizeMatrix = rows * cols * sizeof(float);
    int sizeVec = cols * sizeof(float);

    dim3 dimGrid(ceil(cols/32.0), ceil(rows/32.0), 1)
    dim3 dimBlock(32.0, 32.0, 1)

    matvec_kernel<<<dimGrid, dimBlock>>>(d_Matrix, d_x, d_p, hiddenLayers[i].input, hiddenLayers[i].output)

    for(int i = 0; i < numHidden; i++)
    {
        int sizeWeights = hiddenLayers[i].input*hiddenLayers[i].output*sizeof(float);
        int sizeOut = hiddenLayers[i].output*sizeof(float);
        int sizeIn = hiddenLayers[i].input*sizeof(float);

        CHECK_ERROR(hipMalloc((**void)&d_M, sizeWeights));
        CHECK_ERROR(hipMalloc((**void)&d_x, sizeIn));
        CHECK_ERROR(hipMalloc((**void)&d_p, sizeOut));

        cudaMemCpy(hiddenLayers[i].weights, d_M, sizeWeights, cudaMemCpyHostToDevice);

        if(i == 0){
            cudaMemCpy(inputRow, d_x, sizeIn, cudaMemCpyHostToDevice);
        } else cudaMemCpy(hiddenLayers[i-1].neurons, d_x, sizeIn, cudaMemCpyHostToDevice);

        matvec_kernel<<<dimGrid, dimBlock>>>(d_M, d_x, d_p, hiddenLayers[i].input, hiddenLayers[i].output, true)

        if(i+1 == numHidden) {cudaMemCpy(d_p, yhat, sizeOut, cudaMemCpyDeviceToHost);
        } else cudaMemCpy(d_p, hiddenLayers[i].neurons, sizeOut, cudaMemCpyDeviceToHost);

        hipFree(d_M);
        hipFree(d_x);
        hipFree(d_p);

    }
}

void backpass(layer *hiddenLayers[], float *yhat, float alpha)
{
  //code clearly needed here
}