#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../include/nnKernel.cuh"

#define TILE_SIZE 32
#define CHECK_ERROR(call) { \
	hipError_t err = call; \
	if (err != hipSuccess) { \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(err); \
	} \
}

__global__ void vecxvec_kernel(const float* __restrict__ d_m, const float* __restrict__ d_x, float * __restrict__ d_p,
    const unsigned int nRows, const unsigned int nCols)
{
    const unsigned int tid = blockDimx.x * blockIdx.x + threadIdx.x;
    __shared__ float xds[TILE_SIZE];
    float pval = 0.0;

    #pragma unroll
    for(unsigned int m = 0; m < ((nCols + TILE_SIZE -1)/TILE_SIZE); m++)
    {
        if((m * TILE_SIZE + threadIdx.x) < nCols)
        {
            xds[threadIdx.x] = d_x[threadIdx.x + m * TILE_SIZE];
        } else xds[threadIdx.x] = 0.f;
    }
    __syncthreads();

    if ((Row<Width) && (Col<Width)) p[tid] = Pvalue;

    #pragma unroll
    for(unsigned int e = 0; e < TILE_SIZE; e++)
    {
        pval += d_m[t + (e + TILE_SIZE *m) * nRows] * xds[e];
    }
    
    if ((Row<Width) && (Col<Width)) P[Row*Width+Col] = Pvalue;
}


__global__ void matvec_kernel(const float* __restrict__ d_M, const float* __restrict__ d_x, float * __restrict__ d_p, 
    const unsigned int nRows, const unsigned int nCols, bool ReLU)
{
    const unsigned int tid = blockDimx.x * blockIdx.x + threadIdx.x;
    __shared__ float xds[TILE_SIZE];
    float pval = 0.0;

    #pragma unroll
    for(unsigned int m = 0; m < ((nCols + TILE_SIZE -1)/TILE_SIZE); m++)
    {
        if((m * TILE_SIZE + threadIdx.x) < nCols)
        {
            xds[threadIdx.x] = d_x[threadIdx.x + m * TILE_SIZE];
        } else xds[threadIdx.x] = 0.f;
    
        __syncthreads();

        #pragma unroll
        for(unsigned int e = 0; e < TILE_SIZE; e++)
        {
            pval += d_M[tid + (e + TILE_SIZE *m) * nRows] * xds[e];
        }
    }
    if(ReLU)
    {
        if(t < nRows && pval > 0) d_p[tid] = pval;
    } else{
        if(t < nRows) d_p[tid] = pval;
    }
}

void forwardPass(float *inputRow, layer hiddenLayers[],
    float *yhat, int numHidden)
{
    float *d_Out;
    float *d_Vec;
    float *d_Matrix;
    int sizeMatrix = rows * cols * sizeof(float);
    int sizeVec = cols * sizeof(float);

    dim3 dimGrid(ceil(cols/32.0), ceil(rows/32.0), 1)
    dim3 dimBlock(32.0, 32.0, 1)

    for(int i = 0; i < numHidden; i++)
    {
        int sizeWeights = hiddenLayers[i].input*hiddenLayers[i].output*sizeof(float);
        int sizeOut = hiddenLayers[i].output*sizeof(float);
        int sizeIn = hiddenLayers[i].input*sizeof(float);

        CHECK_ERROR(hipMalloc((**void)&d_M, sizeWeights));
        CHECK_ERROR(hipMalloc((**void)&d_x, sizeIn));
        CHECK_ERROR(hipMalloc((**void)&d_p, sizeOut));

        cudaMemCpy(hiddenLayers[i].weights, d_M, sizeWeights, cudaMemCpyHostToDevice);

        if(i == 0){
            cudaMemCpy(inputRow, d_x, sizeIn, cudaMemCpyHostToDevice);
        } else cudaMemCpy(hiddenLayers[i-1].neurons, d_x, sizeIn, cudaMemCpyHostToDevice);

        matvec_kernel<<<dimGrid, dimBlock>>>(d_M, d_x, d_p, hiddenLayers[i].input, hiddenLayers[i].output, true)

        if(i+1 == numHidden) {cudaMemCpy(d_p, yhat, sizeOut, cudaMemCpyDeviceToHost);
        } else cudaMemCpy(d_p, hiddenLayers[i].neurons, sizeOut, cudaMemCpyDeviceToHost);

        hipFree(d_M);
        hipFree(d_x);
        hipFree(d_p);

    }
}

void backpass(layer *hiddenLayers[], float *yhat, float alpha)
{
  //code clearly needed here
}